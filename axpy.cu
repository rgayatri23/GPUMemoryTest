#include "hip/hip_runtime.h"

/*
 * AXPY :
 * Y += A*X
 */

//#include <iostream>
#include <strings.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <sys/time.h>
#include <nvToolsExt.h>
#if USE_TIMEMORY
#include <timemory/timemory.hpp>
#endif
#include <sys/time.h>

using namespace std;

#define check_num_values 10
#define NUM_LOOPS 10
bool print_csv = false;

//Handle 2-dim pointing
#define Y(i,j) Y[i*M + j]
#define X(i,j) X[i*M + j]

#define dataType double
#define CUDA_HOSTDEV __host__ __device__

inline double elapsedTime(timeval start_time, timeval end_time)
{
  return ((end_time.tv_sec - start_time.tv_sec) +1e-6*(end_time.tv_usec - start_time.tv_usec));
}


void PreComputeFewValues(int N, int M, dataType a, dataType rand1, dataType *Y, dataType *X)
{
  if(N >= check_num_values)
  {
    for(int i = 0; i < check_num_values; ++i)
      for(int j = 0; j < M; ++j)
          X(i,j) = rand1 * (i+1);

    for(int i = 0; i < check_num_values; ++i)
      for(int j = 0; j < M; ++j)
        Y(i,j) += a * X(i,j);
  }
}

void checkGPUCorrectness(int N, int M, dataType *Y, dataType *yOrig)
{
    dataType error = 0.0;
    if(N >= check_num_values)
        for(int i = 0; i < check_num_values; ++i)
        {
            dataType diff = yOrig[i] - Y[i];
            error += diff*diff;
        }

    if(error < 0.0001)
        cout << "Successfull completion of GPU kernels with expected output\n" << endl;
    else
        cout << "Unsuccessfull completion of GPU kernels with diverging results\n" << endl;
}

__global__ void axpyKernel(int N, int M, dataType a, dataType *Y, dataType *X)
{
    for(int i = blockIdx.x; i < N; i+=gridDim.x)
        for(int j = threadIdx.x; j < M; j+=blockDim.x)
            Y(i,j) += a * X(i,j);
}

void zero_copy(dataType a, dataType rand1, double& elapsed_memAlloc, double& elapsed_memcpy, double& elapsed_init, double& elapsed_kernel, int N, int M, dataType* yOrig)
{
  nvtxRangePushA("Zero_copy");
  int device;
  checkCudaErrors(hipSetDevice(3));
  checkCudaErrors(hipGetDevice(&device));
  timeval startMemAllocTimer, endMemAllocTimer,
          startInitTimer, endInitTimer,
          startKernelTimer, endKernelTimer;

  dataType *Y, *X;
  dataType *d_Y, *d_X;
  dim3 grid(N,1,1);
  dim3 threads(32,1,1);

  gettimeofday(&startMemAllocTimer, NULL);
  checkCudaErrors(hipHostAlloc(&X, N*M*sizeof(dataType), hipHostMallocDefault));
  checkCudaErrors(hipHostAlloc(&Y, N*M*sizeof(dataType), hipHostMallocDefault));
  checkCudaErrors(hipHostGetDevicePointer(&d_X,X,0));
  checkCudaErrors(hipHostGetDevicePointer(&d_Y,Y,0));
  gettimeofday(&endMemAllocTimer, NULL);

  gettimeofday(&startInitTimer, NULL);
  memset(Y,0,N*M*sizeof(dataType));
  for(int i = 0; i < N; ++i)
      for(int j = 0; j < M; ++j)
          X[i*M + j] = rand1 * (i+1);
  gettimeofday(&endInitTimer, NULL);

  //Run the kernel twice before any of the timings begin to get rid of the initial-hickups
#if !defined(VERIFY_GPU_CORRECTNESS)
  axpyKernel <<<grid,threads>>> (N,M,a,d_Y,d_X);
  axpyKernel <<<grid,threads>>> (N,M,a,d_Y,d_X);
  checkCudaErrors(hipDeviceSynchronize());
#endif

  //Start Kernel Timer
  gettimeofday(&startKernelTimer, NULL);

  //Start actual kernel//Start Kernel Timerpinned
  axpyKernel <<<grid,threads>>> (N,M,a,d_Y,d_X);
  checkCudaErrors(hipDeviceSynchronize());

  //End Kernel Timer
  gettimeofday(&endKernelTimer, NULL);

#if VERIFY_GPU_CORRECTNESS
  cout << "ZERO-COPY : \t" ;
  checkGPUCorrectness(N,M,Y,yOrig);
#endif

#if !defined(ON_SUMMIT)
  checkCudaErrors(hipHostFree(X));
  checkCudaErrors(hipHostFree(Y));
#endif

  elapsed_memAlloc += elapsedTime(startMemAllocTimer, endMemAllocTimer);
  elapsed_init += elapsedTime(startInitTimer, endInitTimer);
  elapsed_kernel += elapsedTime(startKernelTimer, endKernelTimer);
  nvtxRangePop();
}


void managed_memory(dataType a, dataType rand1, double &elapsed_memAlloc, double& elapsed_memcpy, double& elapsed_init, double &elapsed_kernel, int N, int M, dataType *yOrig)
{
  nvtxRangePushA("managed_memory");
  int device;
  checkCudaErrors(hipSetDevice(2));
  checkCudaErrors(hipGetDevice(&device));
  timeval startMemAllocTimer, endMemAllocTimer,
          startInitTimer, endInitTimer,
          startKernelTimer, endKernelTimer;

  dataType *d_Y, *d_X;
  dim3 grid(N,1,1);
  dim3 threads(32,1,1);

  gettimeofday(&startMemAllocTimer, NULL);
  checkCudaErrors(hipMallocManaged(&d_X, N*M*sizeof(dataType)));
  checkCudaErrors(hipMallocManaged(&d_Y, N*M*sizeof(dataType)));
  gettimeofday(&endMemAllocTimer, NULL);

  gettimeofday(&startInitTimer, NULL);
  for(int i = 0; i < N; ++i)
      for(int j = 0; j < M; ++j)
          d_X[i*M + j] = rand1 * (i+1);
  gettimeofday(&endInitTimer, NULL);

#if !defined(VERIFY_GPU_CORRECTNESS)
  //Run the kernel couple of times before the actual timings begins
  axpyKernel <<<grid,threads>>> (N,M,a,d_Y,d_X);
  axpyKernel <<<grid,threads>>> (N,M,a,d_Y,d_X);
  checkCudaErrors(hipDeviceSynchronize());
#endif

  //Start Kernel Timer
  gettimeofday(&startKernelTimer, NULL);

  //Start actual kernel//Start Kernel Timerpinned
  axpyKernel <<<grid,threads>>> (N,M,a,d_Y,d_X);
  checkCudaErrors(hipDeviceSynchronize());

  //End Kernel Timer
  gettimeofday(&endKernelTimer, NULL);

#if VERIFY_GPU_CORRECTNESS
  cout << "MANAGED-MEMORY : \t" ;
  checkGPUCorrectness(N,M,d_Y,yOrig);
#endif

  checkCudaErrors(hipFree(d_X));
  checkCudaErrors(hipFree(d_Y));

  elapsed_memAlloc += elapsedTime(startMemAllocTimer, endMemAllocTimer);
  elapsed_init += elapsedTime(startInitTimer, endInitTimer);
  elapsed_kernel += elapsedTime(startKernelTimer, endKernelTimer);
  nvtxRangePop();
}

void pinned_memory(dataType a, dataType rand1, double &elapsed_memAlloc, double& elapsed_memcpy, double& elapsed_init, double &elapsed_kernel, int N, int M, dataType* yOrig)
{
  nvtxRangePushA("pinned_memory");
  int device;
  checkCudaErrors(hipSetDevice(1));
  checkCudaErrors(hipGetDevice(&device));
  timeval startMemAllocTimer, endMemAllocTimer,
          startInitTimer, endInitTimer,
          startKernelTimer, endKernelTimer, 
          startMemCpyTimer, endMemCpyTimer;

  dataType *Y, *X;
  dataType *d_Y, *d_X;
  dim3 grid(N,1,1);
  dim3 threads(32,1,1);

  gettimeofday(&startMemAllocTimer, NULL);
//  checkCudaErrors(hipHostAlloc(&X, N*M*sizeof(dataType), hipHostMallocDefault));
//  checkCudaErrors(hipHostAlloc(&Y, N*M*sizeof(dataType), hipHostMallocDefault));
  checkCudaErrors(hipHostAlloc(&X, N*M*sizeof(dataType), hipHostMallocMapped));
  checkCudaErrors(hipHostAlloc(&Y, N*M*sizeof(dataType), hipHostMallocMapped));

    //Allocate memory on device
  checkCudaErrors(hipMalloc(&d_X, N*M*sizeof(dataType)));
  checkCudaErrors(hipMalloc(&d_Y, N*M*sizeof(dataType)));
  gettimeofday(&endMemAllocTimer, NULL);

  gettimeofday(&startInitTimer, NULL);
  memset(Y,0,N*M*sizeof(dataType));
  for(int i = 0; i < N; ++i)
      for(int j = 0; j < M; ++j)
          X[i*M + j] = rand1 * (i+1);
  gettimeofday(&endInitTimer, NULL);

  gettimeofday(&startMemCpyTimer, NULL);
  checkCudaErrors(hipMemcpy(d_X, X, N*M*sizeof(dataType), hipMemcpyHostToDevice));
  gettimeofday(&endMemCpyTimer, NULL);
  elapsed_memcpy += elapsedTime(startMemCpyTimer, endMemCpyTimer);

#if !defined(VERIFY_GPU_CORRECTNESS)
  //Run the kernel couple of times before the actual timings begins
  axpyKernel <<<grid,threads>>> (N,M,a,d_Y,d_X);
  axpyKernel <<<grid,threads>>> (N,M,a,d_Y,d_X);
  checkCudaErrors(hipDeviceSynchronize());
#endif

  //Start Kernel Timer
  gettimeofday(&startKernelTimer, NULL);

  //Start actual kernel//Start Kernel Timerpinned
  axpyKernel <<<grid,threads>>> (N,M,a,d_Y,d_X);
  checkCudaErrors(hipDeviceSynchronize());

  //End Kernel Timer
  gettimeofday(&endKernelTimer, NULL);

  gettimeofday(&startMemCpyTimer, NULL);
  checkCudaErrors(hipMemcpy(Y, d_Y, N*M*sizeof(dataType), hipMemcpyDeviceToHost));
  gettimeofday(&endMemCpyTimer, NULL);

#if VERIFY_GPU_CORRECTNESS
  cout << "PINNED-MEMORY : \t" ;
  checkGPUCorrectness(N,M,Y,yOrig);
#endif

#if !defined(ON_SUMMIT)
  checkCudaErrors(hipHostFree(X));
  checkCudaErrors(hipHostFree(Y));
#endif
  checkCudaErrors(hipFree(d_X));
  checkCudaErrors(hipFree(d_Y));

  elapsed_memAlloc += elapsedTime(startMemAllocTimer, endMemAllocTimer);
  elapsed_init += elapsedTime(startInitTimer, endInitTimer);
  elapsed_memcpy += elapsedTime(startMemCpyTimer, endMemCpyTimer);
  elapsed_kernel += elapsedTime(startKernelTimer, endKernelTimer);
  nvtxRangePop();
}

void pageable_host_device_memory(dataType a, dataType rand1, double &elapsed_memAlloc, double& elapsed_memcpy, double& elapsed_init, double &elapsed_kernel, int N, int M, dataType* yOrig)
{
  nvtxRangePushA("pageable_memory");
  int device;
  checkCudaErrors(hipSetDevice(0));
  checkCudaErrors(hipGetDevice(&device));
  timeval startMemAllocTimer, endMemAllocTimer,
          startInitTimer, endInitTimer,
          startKernelTimer, endKernelTimer, 
          startMemCpyTimer, endMemCpyTimer;

  dataType *Y, *X;
  dataType *d_Y, *d_X;
  dim3 grid(N,1,1);
  dim3 threads(32,1,1);

  gettimeofday(&startMemAllocTimer, NULL);
  X = (dataType*) malloc(N*M*sizeof(dataType));
  Y = (dataType*) malloc(N*M*sizeof(dataType));

  //Allocate memory on device
  checkCudaErrors(hipMalloc(&d_X, N*M*sizeof(dataType)));
  checkCudaErrors(hipMalloc(&d_Y, N*M*sizeof(dataType)));
  gettimeofday(&endMemAllocTimer, NULL);

  gettimeofday(&startInitTimer, NULL);
  memset(Y,0,N*M*sizeof(dataType));
  for(int i = 0; i < N; ++i)
      for(int j = 0; j < M; ++j)
          X[i*M + j] = rand1 * (i+1);
  gettimeofday(&endInitTimer, NULL);

  gettimeofday(&startMemCpyTimer, NULL);
  checkCudaErrors(hipMemcpy(d_X, X, N*M*sizeof(dataType), hipMemcpyHostToDevice));
  gettimeofday(&endMemCpyTimer, NULL);
  elapsed_memcpy += elapsedTime(startMemCpyTimer, endMemCpyTimer);

  //Run the kernel couple of times before the actual timings begins
#if !defined(VERIFY_GPU_CORRECTNESS)
  axpyKernel <<<grid,threads>>> (N,M,a,d_Y,d_X);
  axpyKernel <<<grid,threads>>> (N,M,a,d_Y,d_X);
  checkCudaErrors(hipDeviceSynchronize());
#endif

  //Start Kernel Timer
  gettimeofday(&startKernelTimer, NULL);

  //Start actual kernel//Start Kernel Timerpinned
  axpyKernel <<<grid,threads>>> (N,M,a,d_Y,d_X);
  checkCudaErrors(hipDeviceSynchronize());

  //End Kernel Timer
  gettimeofday(&endKernelTimer, NULL);

  gettimeofday(&startMemCpyTimer, NULL);
  checkCudaErrors(hipMemcpy(Y, d_Y, N*M*sizeof(dataType), hipMemcpyDeviceToHost));
  gettimeofday(&endMemCpyTimer, NULL);
  elapsed_memcpy += elapsedTime(startMemCpyTimer, endMemCpyTimer);

  elapsed_memAlloc += elapsedTime(startMemAllocTimer, endMemAllocTimer);
  elapsed_init += elapsedTime(startInitTimer, endInitTimer);
  elapsed_memcpy += elapsedTime(startMemCpyTimer, endMemCpyTimer);
  elapsed_kernel += elapsedTime(startKernelTimer, endKernelTimer);

#if VERIFY_GPU_CORRECTNESS
  cout << "PAGEABLE-MEMORY : \t" ;
  checkGPUCorrectness(N,M,Y,yOrig);
#endif

  free(X);
  free(Y);
  checkCudaErrors(hipFree(d_X));
  checkCudaErrors(hipFree(d_Y));
  nvtxRangePop();
}

int main(int argc, char **argv)
{
#if USE_TIMEMORY
  tim::timemory_init(argc, argv);
  tim::settings::timing_precision() = 6;
  tim::settings::timing_scientific() = true;
#endif

  int N = 10, M = 10;
  if(argc > 1)
  {
      if(argc == 3)
      {
          N = atoi(argv[1]);
          M = atoi(argv[2]);
      }
      else if(argc == 4)
      {
          N = atoi(argv[1]);
          M = atoi(argv[2]);
          print_csv = true;
      }
      else
      {
          cout << "Input format is ./axpy.ex N M \n" << endl;
          exit(EXIT_FAILURE);
      }
  }

  timeval startTotalTimer, endTotalTimer;
  gettimeofday(&startTotalTimer, NULL);

  cout << "M = " << M << "\t N = " << N << endl;
  cout << "Total Memory Footprint = " << (double)(M*N*sizeof(dataType)/(1024.0*1024.0*1024.0)) << " GBs" << endl;
  cout << "threadblocks = " << N << "  and data accessed by each threadblock = " << (double)(M*sizeof(double)/1024.0) << " Kb" << endl;

#if USE_TIMEMORY
  using namespace tim::component;
  using auto_tuple_t = tim::auto_tuple<cuda_event, real_clock>;
//    using auto_tuple_t = tim::auto_tuple<real_clock, cpu_clock, cpu_util, peak_rss, cuda_event, system_clock, user_clock, cpu_clock>;
  using comp_tuple_t = typename auto_tuple_t::component_type;
  comp_tuple_t measure("total", true);
  comp_tuple_t data_allocation("data_movement", true);
  measure.start();
  data_allocation.start();
#endif

  dataType a = 0.5;
  dataType rand1 = (dataType)rand() / (dataType)RAND_MAX;

  dataType *yOrig, *X;
#if VERIFY_GPU_CORRECTNESS
  yOrig = (dataType*) malloc(check_num_values*M*sizeof(dataType));
  X = (dataType*) malloc(check_num_values*M*sizeof(dataType));
  PreComputeFewValues(N,M,a,rand1,yOrig,X);
#endif


  //Allocating data
#if defined(USE_HOST_PAGEABLE_AND_DEVICE_MEMORY)
  double elapsed_memAlloc, elapsed_kernel, elapsed_init;
  printf("###############Using HOST_PAGEABLE_AND_DEVICE_MEMORY###############\n");
  pageable_host_device_memory(a,rand1,elapsed_memAlloc,elapsed_init, elapsed_kernel,N,M,yOrig);

#elif defined(USE_PINNED_MEMORY)
  double elapsed_memAlloc, elapsed_kernel, elapsed_init;
  printf("###############Using PINNED_MEMORY###############\n");
  pinned_memory(a,rand1,elapsed_memAlloc,elapsed_init,elapsed_kernel,N,M,yOrig);

#elif defined(USE_MANAGED_MEMORY)
  double elapsed_memAlloc, elapsed_kernel, elapsed_init;
  printf("###############Using MANAGED_MEMORY###############\n");
  managed_memory(a,rand1,elapsed_memAlloc,elapsed_init,elapsed_kernel,N,M,yOrig);

#elif defined(USE_ZERO_COPY)
  double elapsed_memAlloc, elapsed_kernel, elapsed_init;
  printf("###############Using ZERO_COPY###############\n");
  zero_copy(a,rand1,elapsed_memAlloc,elapsed_init,elapsed_kernel,N,M,yOrig);

  //Run all the kernels
#elif defined(RUN_ALL)
  printf("###############Running All kernels###############\n");
  double pageable_elapsed_memAlloc = 0.0, pageable_elapsed_kernel = 0.0, pageable_init, pageable_memcpy = 0.0,
         managed_elapsed_memAlloc = 0.0, managed_elapsed_kernel = 0.0, managed_init = 0.0, managed_memcpy = 0.0,
         pinned_elapsed_memAlloc = 0.0, pinned_elapsed_kernel = 0.0, pinned_init = 0.0, pinned_memcpy = 0.0,
         zero_elapsed_memAlloc = 0.0, zero_elapsed_kernel = 0.0, zero_init = 0.0, zero_memcpy = 0.0;

#if !defined(VERIFY_GPU_CORRECTNESS)
  //Run the job for NUM_LOOPS number of times 
  for(int iter = 0; iter < NUM_LOOPS; ++iter) 
#endif
  {
    pageable_host_device_memory(a, rand1, pageable_elapsed_memAlloc, pageable_memcpy, pageable_init, pageable_elapsed_kernel, N, M, yOrig);
    pinned_memory(a, rand1, pinned_elapsed_memAlloc, pinned_memcpy, pinned_init, pinned_elapsed_kernel, N, M, yOrig);
    managed_memory(a, rand1, managed_elapsed_memAlloc, managed_memcpy, managed_init, managed_elapsed_kernel, N, M, yOrig);
    zero_copy(a, rand1, zero_elapsed_memAlloc, zero_memcpy, zero_init, zero_elapsed_kernel, N, M, yOrig);
  }

#if !defined(VERIFY_GPU_CORRECTNESS)
  //Take the average time for each of the runs
    pageable_elapsed_memAlloc /= NUM_LOOPS; pageable_elapsed_kernel /= NUM_LOOPS; pageable_init /= NUM_LOOPS; pageable_memcpy /= NUM_LOOPS;
    managed_elapsed_memAlloc /= NUM_LOOPS; managed_elapsed_kernel /= NUM_LOOPS; managed_init /= NUM_LOOPS; managed_memcpy /= NUM_LOOPS;
    pinned_elapsed_memAlloc /= NUM_LOOPS; pinned_elapsed_kernel /= NUM_LOOPS; pinned_init /= NUM_LOOPS; pinned_memcpy /= NUM_LOOPS;
    zero_elapsed_memAlloc /= NUM_LOOPS; zero_elapsed_kernel /= NUM_LOOPS; zero_init /= NUM_LOOPS; zero_memcpy /= NUM_LOOPS;
#endif

  hipDeviceSynchronize();
#endif

#if USE_TIMEMORY
  data_allocation.stop();
#endif

#if USE_TIMEMORY
    measure.stop();
#endif
  gettimeofday(&endTotalTimer, NULL);

  //calculate elapsed time
  double elapsed_total = elapsedTime(startTotalTimer, endTotalTimer);

#if RUN_ALL
  if(print_csv)
  {
    fprintf(stderr, "Device, \t Memory-Type, \t MemAlloc-time[sec], \t MemCPY-time[sec], \t Kernel-time[sec], \t Kernel+MemAlloc[sec], \t Init-Values[sec]\n");
    fprintf(stderr, "0, \t pageable, \t %f, \t\t %f, \t\t %f, \t\t %f, \t\t %f\n", pageable_elapsed_memAlloc, pageable_memcpy, pageable_elapsed_kernel, pageable_elapsed_memAlloc+pageable_elapsed_kernel, pageable_init);
    fprintf(stderr, "1, \t host-pinned, \t %f, \t\t %f, \t\t %f, \t\t %f, \t\t %f, \n", pinned_elapsed_memAlloc, pinned_memcpy, pinned_elapsed_kernel, pinned_elapsed_memAlloc+pinned_elapsed_kernel,pinned_init);
    fprintf(stderr, "2, \t managed, \t %f, \t\t %f, \t\t %f, \t\t %f, \t\t %f, \n", managed_elapsed_memAlloc, managed_memcpy, managed_elapsed_kernel, managed_elapsed_memAlloc+managed_elapsed_kernel, managed_init);
    fprintf(stderr, "3, \t zero-copy, \t %f, \t\t %f, \t\t %f, \t\t %f, \t\t %f \n", zero_elapsed_memAlloc, zero_memcpy, zero_elapsed_kernel, zero_elapsed_memAlloc+zero_elapsed_kernel, zero_init);
  }
  else
  {
    fprintf(stderr, "-------------------------------------------------------------------------------------------------------------------------------------------\n");
    fprintf(stderr, "Device, \t Memory-Type, \t MemAlloc-time[sec], \t MemCPY-time[sec], \t Kernel-time[sec], \t Kernel+MemAlloc[sec], \t Init-Values[sec]\n");
    fprintf(stderr, "-------------------------------------------------------------------------------------------------------------------------------------------\n");
    fprintf(stderr, "0 \t pageable, \t %f, \t\t %f, \t\t %f, \t\t %f, \t\t %f\n", pageable_elapsed_memAlloc, pageable_memcpy, pageable_elapsed_kernel, pageable_elapsed_memAlloc+pageable_elapsed_kernel, pageable_init);
    fprintf(stderr, "-------------------------------------------------------------------------------------------------------------------------------------------\n");
    fprintf(stderr, "1 \t host-pinned, \t %f, \t\t %f, \t\t %f, \t\t %f, \t\t %f, \n", pinned_elapsed_memAlloc, pinned_memcpy, pinned_elapsed_kernel, pinned_elapsed_memAlloc+pinned_elapsed_kernel,pinned_init);
    fprintf(stderr, "-------------------------------------------------------------------------------------------------------------------------------------------\n");
    fprintf(stderr, "2 \t managed, \t %f, \t\t %f, \t\t %f, \t\t %f, \t\t %f, \n", managed_elapsed_memAlloc, managed_memcpy, managed_elapsed_kernel, managed_elapsed_memAlloc+managed_elapsed_kernel, managed_init);
    fprintf(stderr, "-------------------------------------------------------------------------------------------------------------------------------------------\n");
    fprintf(stderr, "3 \t zero-copy, \t %f, \t\t %f, \t\t %f, \t\t %f, \t\t %f \n", zero_elapsed_memAlloc, zero_memcpy, zero_elapsed_kernel, zero_elapsed_memAlloc+zero_elapsed_kernel, zero_init);
    fprintf(stderr, "-------------------------------------------------------------------------------------------------------------------------------------------\n");
  }

  cout << "************ Total-time = " << elapsed_total << " [sec] ************\n" << endl;

#else
  cout << "************ MemAlloc-time = " << elapsed_memAlloc << " [sec] ************\n" << endl;
  cout << "************ Kernel-time = " << elapsed_kernel << " [sec] ************\n" << endl;
  cout << "************ Total-time = " << elapsed_total << " [sec] ************\n" << endl;
#endif
    return 0;
}
