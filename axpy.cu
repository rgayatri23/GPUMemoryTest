#include "hip/hip_runtime.h"

/*
 * AXPY :
 * Y += A*X
 */

//#include <iostream>
#include <strings.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <sys/time.h>
#if USE_TIMEMORY
#include <timemory/timemory.hpp>
#endif
#include <sys/time.h>

using namespace std;

#define check_num_values 10

//Handle 2-dim pointing
#define Y(i,j) Y[i*M + j]
#define X(i,j) X[i*M + j]

#define dataType double
#define CUDA_HOSTDEV __host__ __device__

inline double elapsedTime(timeval start_time, timeval end_time)
{
  return ((end_time.tv_sec - start_time.tv_sec) +1e-6*(end_time.tv_usec - start_time.tv_usec));
}


void PreComputeFewValues(int N, int M, dataType a, dataType *Y, dataType *X)
{
  if(N >= check_num_values)
    for(int i = 0; i < check_num_values; ++i)
        for(int j = 0; j < M; ++j)
            Y(i,j) += a * X(i,j);
}

void checkGPUCorrectness(int N, int M, dataType *Y, dataType *yOrig)
{
    dataType error = 0.0;
    if(N >= check_num_values)
        for(int i = 0; i < check_num_values; ++i)
        {
            dataType diff = yOrig[i] - Y[i];
            error += diff*diff;
        }

    if(error < 0.0001)
        cout << "Successfull completion of GPU kernels with expected output\n" << endl;
    else
        cout << "Unsuccessfull completion of GPU kernels with diverging results\n" << endl;
}

__global__ void axpyKernel(int N, int M, dataType a, dataType *Y, dataType *X)
{
    for(int i = blockIdx.x; i < N; i+=gridDim.x)
        for(int j = threadIdx.x; j < M; j+=blockDim.x)
            Y(i,j) += a * X(i,j);
}

int main(int argc, char **argv)
{
#if USE_TIMEMORY
  tim::timemory_init(argc, argv);
  tim::settings::timing_precision() = 6;
  tim::settings::timing_scientific() = true;
#endif

  int N = 10, M = 10;
  if(argc > 1)
  {
      if(argc == 3)
      {
          N = atoi(argv[1]);
          M = atoi(argv[2]);
      }
      else
      {
          cout << "Input format is ./axpy.ex N M \n" << endl;
          exit(EXIT_FAILURE);
      }
  }

  double elapsed_total, elapsed_memAlloc, elapsed_kernel;
  timeval startTotalTimer, endTotalTimer,
          startMemAllocTimer, endMemAllocTimer,
          startKernelTimer, endKernelTimer;
  gettimeofday(&startTotalTimer, NULL);

  cout << "M = " << M << "\t N = " << N << endl;
  cout << "Total Memory Footprint = " << (size_t)(M*N*sizeof(dataType)/(1024*1024*1024)) << " GBs" << endl;
  cout << "threadblocks = " << N << "  and data accessed by each threadblock = " << M*sizeof(double) << " bytes" << endl;

  int device;
//    checkCudaErrors(hipSetDevice(1));
  checkCudaErrors(hipGetDevice(&device));
  cout << "Device number = " << device << endl;

  dataType *Y, *X;
  dataType *d_Y, *d_X;
  bool copyFlag = false;

  dim3 grid(N,1,1);
  dim3 threads(M,1,1);

  dataType a = 0.5;
  dataType rand1 = (dataType)rand() / (dataType)RAND_MAX;

#if USE_TIMEMORY
  using namespace tim::component;
  using auto_tuple_t = tim::auto_tuple<cuda_event, real_clock>;
//    using auto_tuple_t = tim::auto_tuple<real_clock, cpu_clock, cpu_util, peak_rss, cuda_event, system_clock, user_clock, cpu_clock>;
  using comp_tuple_t = typename auto_tuple_t::component_type;
  comp_tuple_t measure("total", true);
  comp_tuple_t data_allocation("data_movement", true);
  measure.start();
  data_allocation.start();
#endif

  //Allocating data
  gettimeofday(&startMemAllocTimer, NULL);
#if defined(USE_HOST_PAGEABLE_AND_DEVICE_MEMORY)
  printf("###############Using HOST_PAGEABLE_AND_DEVICE_MEMORY###############\n");
  X = (dataType*) malloc(N*M*sizeof(dataType));
  Y = (dataType*) malloc(N*M*sizeof(dataType));

  //Allocate memory on device
  checkCudaErrors(hipMalloc(&d_X, N*M*sizeof(dataType)));
  checkCudaErrors(hipMalloc(&d_Y, N*M*sizeof(dataType)));

  copyFlag = true; //Switch on the copy flag

#elif defined(USE_PINNED_MEMORY)
  printf("###############Using PINNED_MEMORY###############\n");
  checkCudaErrors(hipHostMalloc(&X, N*M*sizeof(dataType)));
  checkCudaErrors(hipHostMalloc(&Y, N*M*sizeof(dataType)));

    //Allocate memory on device
  checkCudaErrors(hipMalloc(&d_X, N*M*sizeof(dataType)));
  checkCudaErrors(hipMalloc(&d_Y, N*M*sizeof(dataType)));

  copyFlag = true; //Switch on the copy flag

#elif defined(USE_MANAGED_MEMORY)
  printf("###############Using MANAGED_MEMORY###############\n");
  checkCudaErrors(hipMallocManaged(&d_X, N*M*sizeof(dataType)));
  checkCudaErrors(hipMallocManaged(&d_Y, N*M*sizeof(dataType)));
  X = d_X; Y = d_Y;

#elif defined(USE_ZERO_COPY)
  printf("###############Using ZERO_COPY###############\n");
  checkCudaErrors(hipHostMalloc(&X, N*M*sizeof(dataType)));
  checkCudaErrors(hipHostMalloc(&Y, N*M*sizeof(dataType)));
  checkCudaErrors(hipHostGetDevicePointer(&d_X,X,0));
  checkCudaErrors(hipHostGetDevicePointer(&d_Y,Y,0));
#endif
  gettimeofday(&endMemAllocTimer, NULL);

  memset(Y,0,N*M*sizeof(dataType));
  for(int i = 0; i < N; ++i)
      for(int j = 0; j < M; ++j)
          X[i*M + j] = rand1 * (i+1);

  if(copyFlag == true)
      checkCudaErrors(hipMemcpy(d_X, X, N*M*sizeof(dataType), hipMemcpyHostToDevice));

#if VERIFY_GPU_CORRECTNESS
  dataType *yOrig;
  yOrig = (dataType*) malloc(check_num_values*M*sizeof(dataType));
  PreComputeFewValues(N,M,a,yOrig,X);
#endif

  //Actual CUDA kernel
  gettimeofday(&startKernelTimer, NULL);
  axpyKernel <<<grid,threads>>> (N,M,a,d_Y,d_X);

  checkCudaErrors(hipDeviceSynchronize());
  if(copyFlag)
      checkCudaErrors(hipMemcpy(Y, d_Y, N*M*sizeof(dataType), hipMemcpyDeviceToHost));
  gettimeofday(&endKernelTimer, NULL);

#if USE_TIMEMORY
  data_allocation.stop();
#endif


#if VERIFY_GPU_CORRECTNESS
  checkGPUCorrectness(N,M,Y,yOrig);
#endif

#if defined (USE_HOST_PAGEABLE_AND_DEVICE_MEMORY)
    free(X);
    free(Y);
    checkCudaErrors(hipFree(d_X));
    checkCudaErrors(hipFree(d_Y));
#elif defined(USE_PINNED_MEMORY)
    checkCudaErrors(hipHostFree(X));
    checkCudaErrors(hipHostFree(Y));
    checkCudaErrors(hipFree(d_X));
    checkCudaErrors(hipFree(d_Y));
#elif defined(USE_MANAGED_MEMORY)
    checkCudaErrors(hipFree(d_X));
    checkCudaErrors(hipFree(d_Y));
#elif defined(USE_ZERO_COPY)
    checkCudaErrors(hipHostFree(X));
    checkCudaErrors(hipHostFree(Y));
#endif

#if USE_TIMEMORY
    measure.stop();
#endif
  gettimeofday(&endTotalTimer, NULL);

  //calculate elapsed time
  elapsed_total = elapsedTime(startTotalTimer, endTotalTimer);
  elapsed_memAlloc = elapsedTime(startMemAllocTimer, endMemAllocTimer);
  elapsed_kernel = elapsedTime(startKernelTimer, endKernelTimer);

  cout << "************ MemAlloc-time = " << elapsed_memAlloc << " [sec] ************\n" << endl;
  cout << "************ Kernel-time = " << elapsed_kernel << " [sec] ************\n" << endl;
  cout << "************ Total-time = " << elapsed_total << " [sec] ************\n" << endl;
    return 0;
}
